#include"header_simulation_engine.h"

#define NUMITERATION 10
#define NUMPROJECTIONS 14 
#define DETECTORWIDTH 100
#define DETECTORHEIGHT 100
#define NUMLOCALX 32
#define NUMLOCALY 32
#define PITCH 0.1

int main(){
	char buffer[50];
	int i, idxProjection, idxImigrationLocal, idxImigrationImage, *imageReal, *imageRealVertical, *imageRealHorizontal, idxIteration, objectPointNum=50*50*50, numLocal=NUMLOCALX*NUMLOCALY, detectorNumWidth=DETECTORWIDTH/PITCH, detectorNumHeight=DETECTORHEIGHT/PITCH, detectorNum=detectorNumWidth*detectorNumHeight, *cor, *cor_dev, numBlockAdjust=(numLocal*NUMPROJECTIONS)/32+1, driftX=detectorNumWidth/2-NUMLOCALX/2, driftY=detectorNumHeight/2-NUMLOCALY/2;
	float *object, *image, *imageVertical, *imageHorizontal, *image_dev, *imageVertical_dev, *imageHorizontal_dev, *psfPartial, *psfPartial_dev, psfBuffer, integrationAdjust, integrationPsf;
	FILE *fidRecon, *fidImageReal, *fidPSF;

	object=(float *)malloc(sizeof(float)*objectPointNum);
	image=(float *)malloc(sizeof(float)*detectorNumWidth*detectorNumHeight*NUMPROJECTIONS);
	imageVertical=(float *)malloc(sizeof(float)*detectorNumWidth*NUMPROJECTIONS);
	imageHorizontal=(float *)malloc(sizeof(float)*detectorNumWidth*NUMPROJECTIONS);
	imageReal=(int *)malloc(sizeof(int)*detectorNumWidth*detectorNumHeight*NUMPROJECTIONS);
	imageRealVertical=(int *)malloc(sizeof(int)*detectorNumWidth*NUMPROJECTIONS);
	imageRealHorizontal=(int *)malloc(sizeof(int)*detectorNumWidth*NUMPROJECTIONS);
	hipMalloc((void**)&image_dev,sizeof(float)*detectorNumWidth*detectorNumHeight*NUMPROJECTIONS);
	hipMalloc((void**)&imageVertical_dev,sizeof(float)*detectorNumWidth*NUMPROJECTIONS);
	hipMalloc((void**)&imageHorizontal_dev,sizeof(float)*detectorNumWidth*NUMPROJECTIONS);
	cor=(int *)malloc(sizeof(int)*2);
	hipMalloc((void**)&cor_dev,sizeof(int)*2);
	psfPartial=(float *)malloc(sizeof(float)*numLocal);
	hipMalloc((void**)&psfPartial_dev,sizeof(float)*numLocal);
	///*
	fidImageReal=fopen("G:\\research\\data\\projectionImage.bin", "rb");
	fread(imageReal, sizeof(int), detectorNumWidth*detectorNumHeight*NUMPROJECTIONS, fidImageReal);
	fclose(fidImageReal);
	//*/
	///*
	fidImageReal=fopen("G:\\research\\data\\projectionImageVerticalSlit.bin", "rb");
	fread(imageRealVertical, sizeof(int), detectorNumWidth*NUMPROJECTIONS, fidImageReal);
	fclose(fidImageReal);

	fidImageReal=fopen("G:\\research\\data\\projectionImageHorizontalSlit.bin", "rb");
	fread(imageRealHorizontal, sizeof(int), detectorNumWidth*NUMPROJECTIONS, fidImageReal);
	fclose(fidImageReal);
	//*/
	for(i=0;i<objectPointNum;i++){
		object[i]=1;
	}
	

	for(idxIteration=0;idxIteration<NUMITERATION;idxIteration++){
		///*
		for(i=0;i<detectorNumWidth*detectorNumHeight*NUMPROJECTIONS;i++){
			image[i]=0;
		}
		hipMemcpy(image_dev, image, sizeof(float)*detectorNumWidth*detectorNumHeight*NUMPROJECTIONS, hipMemcpyHostToDevice);		
		//*/
		///*
		for(i=0;i<detectorNumWidth*NUMPROJECTIONS;i++){
			imageVertical[i]=0;
		}
		hipMemcpy(imageVertical_dev, imageVertical, sizeof(float)*detectorNumWidth*NUMPROJECTIONS, hipMemcpyHostToDevice);
		for(i=0;i<detectorNumWidth*NUMPROJECTIONS;i++){
			imageHorizontal[i]=0;
		}
		hipMemcpy(imageHorizontal_dev, imageHorizontal, sizeof(float)*detectorNumWidth*NUMPROJECTIONS, hipMemcpyHostToDevice);

		forwardProjectionAllPoints(image, image_dev, cor, cor_dev, psfPartial, psfPartial_dev, NUMPROJECTIONS, object, objectPointNum, NUMLOCALX, NUMLOCALY, detectorNumWidth, detectorNumHeight, detectorNum, 'p');
		forwardProjectionAllPoints(imageVertical, imageVertical_dev, cor, cor_dev, psfPartial, psfPartial_dev, NUMPROJECTIONS, object, objectPointNum, NUMLOCALX, NUMLOCALY, detectorNumWidth, detectorNumHeight, detectorNumWidth, 'v');
		forwardProjectionAllPoints(imageHorizontal, imageHorizontal_dev, cor, cor_dev, psfPartial, psfPartial_dev, NUMPROJECTIONS, object, objectPointNum, NUMLOCALX, NUMLOCALY, detectorNumWidth, detectorNumHeight, detectorNumWidth, 'h');

		for(i=0;i<objectPointNum;i++){
			integrationAdjust=0;
			integrationPsf=0;
			for(idxProjection=0;idxProjection<NUMPROJECTIONS;idxProjection++){
				
				sprintf(buffer, "G:\\research\\data\\PSF%d.bin", idxProjection);
				fidPSF=fopen(buffer, "rb");
				fseek(fidPSF, ((numLocal+2)*i)*4, SEEK_SET);
				fread(cor, sizeof(int), 2, fidPSF);
				for(idxImigrationLocal=0;idxImigrationLocal<numLocal;idxImigrationLocal++){
					fread(&psfBuffer, sizeof(float), 1, fidPSF);
					idxImigrationImage=(cor[0]+driftX+idxImigrationLocal/32)*detectorNumHeight+(cor[1]+driftY+idxImigrationLocal%32);
					if(idxImigrationImage>=0&&idxImigrationImage<detectorNum){
						if(image[idxImigrationImage+detectorNum*idxProjection]){
							integrationAdjust=integrationAdjust+imageReal[idxImigrationImage+detectorNum*idxProjection]/image[idxImigrationImage+detectorNum*idxProjection]*psfBuffer;
							integrationPsf=integrationPsf+psfBuffer;
						}
					}
				}
				fclose(fidPSF);
				
				///*
				sprintf(buffer, "G:\\research\\data\\vslitPSF%d.bin", idxProjection);
				fidPSF=fopen(buffer, "rb");
				fseek(fidPSF, ((NUMLOCALX+1)*i)*4, SEEK_SET);
				fread(cor, sizeof(int), 1, fidPSF);
				for(idxImigrationLocal=0;idxImigrationLocal<NUMLOCALX;idxImigrationLocal++){
					fread(&psfBuffer, sizeof(float), 1, fidPSF);
					idxImigrationImage=cor[0]+driftX+idxImigrationLocal;
					if(idxImigrationImage>=0&&idxImigrationImage<detectorNumWidth){
						if(imageVertical[idxImigrationImage+detectorNumWidth*idxProjection]){
							integrationAdjust=integrationAdjust+imageRealVertical[idxImigrationImage+detectorNumWidth*idxProjection]/imageVertical[idxImigrationImage+detectorNumWidth*idxProjection]*psfBuffer;
							integrationPsf=integrationPsf+psfBuffer;
						}
					}
				}
				fclose(fidPSF);

				sprintf(buffer, "G:\\research\\data\\hslitPSF%d.bin", idxProjection);
				fidPSF=fopen(buffer, "rb");
				fseek(fidPSF, ((NUMLOCALX+1)*i)*4, SEEK_SET);
				fread(cor, sizeof(int), 1, fidPSF);
				for(idxImigrationLocal=0;idxImigrationLocal<NUMLOCALX;idxImigrationLocal++){
					fread(&psfBuffer, sizeof(float), 1, fidPSF);
					idxImigrationImage=cor[0]+driftX+idxImigrationLocal;
					if(idxImigrationImage>=0&&idxImigrationImage<detectorNumWidth){
						if(imageHorizontal[idxImigrationImage+detectorNumWidth*idxProjection]){
							integrationAdjust=integrationAdjust+imageRealHorizontal[idxImigrationImage+detectorNumWidth*idxProjection]/imageHorizontal[idxImigrationImage+detectorNumWidth*idxProjection]*psfBuffer;
							integrationPsf=integrationPsf+psfBuffer;
						}
					}
				}
				fclose(fidPSF);
				//*/
			}
			object[i]=object[i]*integrationAdjust/integrationPsf;
		}				
		
		printf("Iteration %d is finished!\n", idxIteration);
		fidRecon=fopen("G:\\research\\data\\recon.bin", "wb");
		fwrite(object, sizeof(float), objectPointNum, fidRecon);
		fclose(fidRecon);
	}
	fidRecon=fopen("G:\\research\\data\\recon.bin", "wb");
	fwrite(object, sizeof(float), objectPointNum, fidRecon);
	fclose(fidRecon);
	getchar();
}