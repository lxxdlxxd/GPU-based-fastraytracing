#include "hip/hip_runtime.h"
#include"header_simulation_engine.h"

int main(){
	int numProjection, numFacelet1, numFacelet2, numStack=0, i, j, k;
	specFacet* part1,* part2, *stack, *stackRotatedToVector, *stackRotatedToVector_dev;

	readPart(&part1, &numFacelet1, "G:\\research\\slitSPECT\\pinholeAperture_assemble_brain_projection_1.5mm_2.25mm - pinholeAperture-1.STL",'t');
	readPart(&part2, &numFacelet2, "G:\\research\\slitSPECT\\pinholeAperture_assemble_brain_projection_1.5mm_2.25mm - detector-1.STL",'d');
	catenate(&stack, &numStack, &part1, & numFacelet1);
	catenate(&stack, &numStack, &part2, & numFacelet2);

	stackRotatedToVector=(specFacet *)malloc(numStack*sizeof(specFacet));

	float M[9], *nv_dev, photonPosition[3], *photonPosition_dev, EdgePointFinal[3], EdgePointOriginal[3], rotationSpec[2];
	float MBack[9], *nv, *M_dev;
	float centerOfAperturePosition[3], alpha, directionVector[15][3];
	double *randomArray_dev;
	char fileName[50];
	int numPhoton=0;
	short *phantom, value;
	unsigned long int accumulatedNum=0;
	char *workPlace, *workPlace_dev;
	vectors *vectorsResult_dev;
	hiprandState *state_dev;
	FILE *fid, *fidPhantom;
	std::default_random_engine generator;

	nv=(float *)malloc(sizeof(float)*NUMPERBATCH*3);
	hipMalloc((void**)&nv_dev, sizeof(float)*NUMPERBATCH*3);
	hipMalloc((void**)&state_dev, sizeof(hiprandState)*NUMPERBATCH*2);
	hipMalloc((void**)&randomArray_dev, sizeof(float)*NUMPERBATCH*2);
	hipMalloc((void**)&photonPosition_dev, sizeof(float)*3);
	hipMalloc((void**)&stackRotatedToVector_dev, sizeof(specFacet)*numStack);
	workPlace=(char *)malloc(sizeof(char)*numStack*NUMPERBATCH);
	hipMalloc((void**)&workPlace_dev,sizeof(char)*numStack*NUMPERBATCH);
	hipMalloc((void**)&vectorsResult_dev, sizeof(vectors)*numStack);
	hipMalloc((void**)&M_dev, sizeof(float)*9);
	
	phantom=(short *)malloc(sizeof(short)*181*217*181);
	fidPhantom=fopen("G:\\research\\numericalPhantom\\phantom_1.0mm_normal_gry.raws", "rb");
	fread(phantom, sizeof(short), 181*217*181, fidPhantom);
	fclose(fidPhantom);

	std::clock_t start, end;
	start=std::clock();

	directionVector[0][0]=-125.8;
	directionVector[0][1]=-139.3;
	directionVector[0][2]=-71.69;

	directionVector[1][0]=-163.84;
	directionVector[1][1]=-8.34;
	directionVector[1][2]=-116;

	directionVector[2][0]=-187.35;
	directionVector[2][1]=72.59;
	directionVector[2][2]=0;

	directionVector[3][0]=-163.84;
	directionVector[3][1]=-8.34;
	directionVector[3][2]=116;

	directionVector[4][0]=-125.8;
	directionVector[4][1]=-139.3;
	directionVector[4][2]=71.69;

	directionVector[5][0]=8.34;
	directionVector[5][1]=-163.84;
	directionVector[5][2]=-116;

	directionVector[6][0]=8.34;
	directionVector[6][1]=-163.84;
	directionVector[6][2]=116;

	directionVector[7][0]=-53.21;
	directionVector[7][1]=48.05;
	directionVector[7][2]=187.7;

	directionVector[8][0]=-91.25;
	directionVector[8][1]=179.01;
	directionVector[8][2]=0;

	directionVector[9][0]=-53.21;
	directionVector[9][1]=48.05;
	directionVector[9][2]=-187.7;

	directionVector[10][0]=53.21;
	directionVector[10][1]=-48.05;
	directionVector[10][2]=-187.7;

	directionVector[11][0]=91.25;
	directionVector[11][1]=-179.01;
	directionVector[11][2]=0;

	directionVector[12][0]=53.21;
	directionVector[12][1]=-48.05;
	directionVector[12][2]=187.7;

	directionVector[13][0]=-8.34;
	directionVector[13][1]=163.84;
	directionVector[13][2]=116;

	directionVector[14][0]=-8.34;
	directionVector[14][1]=163.84;
	directionVector[14][2]=-116;

	for(numProjection=0;numProjection<1;numProjection++){

		sprintf(fileName,"G:\\research\\data\\listModeProjection%d_test.bin",numProjection);
		fid=fopen(fileName,"wb");
		centerOfAperturePosition[0]=directionVector[numProjection][0];
		centerOfAperturePosition[1]=directionVector[numProjection][1];
		centerOfAperturePosition[2]=directionVector[numProjection][2];
		alpha=0;
		rotateFacetToVector(stackRotatedToVector, stack, numStack, alpha, centerOfAperturePosition);
		hipMemcpy(stackRotatedToVector_dev, stackRotatedToVector, sizeof(specFacet)*numStack, hipMemcpyHostToDevice);
		EdgePointOriginal[0]=0;
		EdgePointOriginal[1]=1;
		EdgePointOriginal[2]=0;
		rotationFacetEdgePointGeneration(MBack, EdgePointFinal, EdgePointOriginal, alpha, centerOfAperturePosition);

		for(i=0;i<181;i++){
			for(j=0;j<217;j++){
				for(k=0;k<181;k++){
					photonPosition[0]=i-90;
					photonPosition[1]=j-108;
					photonPosition[2]=k-90;
					rotationAngleGeneration(rotationSpec, EdgePointFinal, photonPosition, centerOfAperturePosition);
					value=phantom[217*181*i+181*j+k];
					if(value){
						std::poisson_distribution<int> distributionP(value);
						numPhoton=distributionP(generator);
					}else{
						numPhoton=0;
					}
					
					velocityRotationMatrixGeneration(M, rotationSpec, photonPosition, centerOfAperturePosition, 0);
					hipMemcpy(photonPosition_dev, photonPosition, sizeof(float)*3, hipMemcpyHostToDevice);
					hipMemcpy(M_dev, M, sizeof(float)*9, hipMemcpyHostToDevice);
					//fastRayTracing(detector, DETECTORWIDTH, DETECTORHEIGHT, PITCH, nv, nv_dev, randomArray_dev, state_dev, numPhoton, photonPosition, photonPosition_dev, stackRotatedToVector, stackRotatedToVector_dev, numStack, workPlace, workPlace_dev, vectorsResult_dev, &accumulatedNum, M_dev, MBack);
					if(numPhoton){
						fastRayTracingListMode(fid, nv, nv_dev, randomArray_dev, state_dev, numPhoton, photonPosition, photonPosition_dev, stackRotatedToVector, stackRotatedToVector_dev, numStack, workPlace, workPlace_dev, vectorsResult_dev, &accumulatedNum, M_dev, MBack);
					}
				}
			}
			printf("%d is complete\n", i);
		}

		fclose(fid);
		printf("Projection %d is completed!!\n", numProjection);
	}	
	end=std::clock();		
	
	printf("time is %d\n", end-start);
	getchar();
}