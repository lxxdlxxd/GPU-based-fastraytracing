#include "hip/hip_runtime.h"
#include"header_simulation_engine.h"

#define NUMPROJECTIONS 90
#define QUEUELENGTH 10							//the size of maximum number of facelets a photon traveling in a straight line can pass
#define DEVIDENT 8								//The striding step in search the result matrix workplace, in order to search faster					
#define NUMPERBATCH 1024
#define NUMVECTORBIN 600
#define PITCHBIN 0.1
#define NUMANGLE 1
#define PHANTOMSIZEX 50
#define PHANTOMSIZEY 50
#define PHANTOMSIZEZ 50
#define PHANTOMPITCHX 0.5
#define PHANTOMPITCHY 0.5
#define PHANTOMPITCHZ 0.5

int main(){
	char *workPlace, *workPlace_dev, buffer[150];
	int idxAngle, idxProjection, value, numFacelet1, numFacelet2, numStack=0, i, j, k, cors[2], numMax=0, *vectorBin;
	float theta, phi, valueFloat, M_source[9], M_velocity[9], *M_velocity_dev, *nv, *nv_dev, photonPosition[3], tempPosition[3], centerOfAperturePosition[3], *photonPosition_dev, angleAlpha=0, distOriginToCA=51.5, rotationSpec[2], EdgePointFinal[3]={0,1,0}, *phantom;
	double *randomArray_dev;
	unsigned long int accumulatedNum=0;
	specFacet* part1,* part2, *stack, *stack_dev;
	vectors *vectorsResult_dev;
	FILE *fid, *fidPhantom;
	std::default_random_engine generator;
	hiprandState *state_dev;

	readPart(&part1, &numFacelet1, "G:\\research\\slitSPECT\\slitAperture_assemble50X50_spacing_0.3 - slitAperture50X50_0.3spacing-1.STL",'t');
	readPart(&part2, &numFacelet2, "G:\\research\\slitSPECT\\slitAperture_assemble50X50_spacing_0.3 - detector50X50-1.STL",'d');                
	catenate(&stack, &numStack, &part1, & numFacelet1);
	catenate(&stack, &numStack, &part2, & numFacelet2);

	hipMalloc((void**)&stack_dev,sizeof(specFacet)*numStack);
	hipMemcpy(stack_dev, stack, sizeof(specFacet)*numStack, hipMemcpyHostToDevice);		
	nv=(float *)malloc(sizeof(float)*NUMPERBATCH*3);
	hipMalloc((void**)&nv_dev, sizeof(float)*NUMPERBATCH*3);
	hipMalloc((void**)&photonPosition_dev, sizeof(float)*3);
	hipMalloc((void**)&M_velocity_dev, sizeof(float)*9);
	workPlace=(char *)malloc(sizeof(char)*numStack*NUMPERBATCH);
	hipMalloc((void**)&workPlace_dev,sizeof(char)*numStack*NUMPERBATCH);
	hipMalloc((void**)&vectorsResult_dev, sizeof(vectors)*numStack);
	vectorBin=(int *)malloc(sizeof(int)*NUMVECTORBIN);
	hipMalloc((void**)&randomArray_dev, sizeof(float)*NUMPERBATCH*2);
	hipMalloc((void**)&state_dev, sizeof(hiprandState)*NUMPERBATCH*2);
	phantom=(float *)malloc(sizeof(float)*PHANTOMSIZEX*PHANTOMSIZEY*PHANTOMSIZEZ);

	fidPhantom=fopen("G:\\research\\data\\phantom.bin","rb");
	fread(phantom, sizeof(float), PHANTOMSIZEX*PHANTOMSIZEY*PHANTOMSIZEZ, fidPhantom);
	fclose(fidPhantom);
	std::clock_t start, end;
	start=std::clock();
	//
	int test;
	//
	for(idxAngle=0;idxAngle<NUMANGLE;idxAngle++){

		theta=PI/2;
		phi=0;

		sprintf(buffer, "G:\\research\\data\\rotatingSlitProjection_resolution_phantomX1.8_90_600X10_0.9A%d_view_45_45_Poisson_withoutAAS.bin", idxAngle);
		fid=fopen(buffer,"wb");

		for(idxProjection=0;idxProjection<NUMPROJECTIONS;idxProjection++){		
			centerOfAperturePosition[0]=distOriginToCA*sin(theta)*cos(phi);//30.75;
			centerOfAperturePosition[1]=distOriginToCA*sin(theta)*sin(phi);//30.75;
			centerOfAperturePosition[2]=distOriginToCA*cos(theta);//43.487067043;

			angleAlpha=PI*idxProjection/NUMPROJECTIONS;

			inverseMatrixGeneration(M_source, angleAlpha, centerOfAperturePosition);

			centerOfAperturePosition[0]=0;
			centerOfAperturePosition[1]=0;
			centerOfAperturePosition[2]=distOriginToCA;

			for(i=0;i<NUMVECTORBIN;i++){
				vectorBin[i]=0;
			}
			/*
			int countTotal=0;
			for (i = 0; i < PHANTOMSIZEX; i++){
				for (j = 0; j < PHANTOMSIZEY; j++){
					for (k = 0; k < PHANTOMSIZEZ; k++){
						countTotal += phantom[i*PHANTOMSIZEY*PHANTOMSIZEZ + j*PHANTOMSIZEZ + k];
					}
				}
			}
			*/
			for(i=0;i<PHANTOMSIZEX;i++){
				for(j=0;j<PHANTOMSIZEY;j++){
					for(k=0;k<PHANTOMSIZEZ;k++){
						valueFloat=phantom[i*PHANTOMSIZEY*PHANTOMSIZEZ+j*PHANTOMSIZEZ+k]*180;//0.6;
						if(valueFloat){
							std::poisson_distribution<int> distributionPo(valueFloat);
							value=distributionPo(generator);
						}else{
							value=0;
						}
						if(value){						

							tempPosition[0]=(i-PHANTOMSIZEX/2.0)*PHANTOMPITCHX;
							tempPosition[1]=(j-PHANTOMSIZEY/2.0)*PHANTOMPITCHY;
							tempPosition[2]=(k-PHANTOMSIZEZ/2.0)*PHANTOMPITCHZ;
							photonPosition[0]=tempPosition[0]*M_source[0]+tempPosition[1]*M_source[1]+tempPosition[2]*M_source[2];
							photonPosition[1]=tempPosition[0]*M_source[3]+tempPosition[1]*M_source[4]+tempPosition[2]*M_source[5];
							photonPosition[2]=tempPosition[0]*M_source[6]+tempPosition[1]*M_source[7]+tempPosition[2]*M_source[8];
							hipMemcpy(photonPosition_dev, photonPosition, sizeof(float)*3, hipMemcpyHostToDevice);

							rotationAngleGeneration(rotationSpec, EdgePointFinal, photonPosition, centerOfAperturePosition);
							velocityRotationMatrixGeneration(M_velocity, rotationSpec, photonPosition, centerOfAperturePosition, 1);
							hipMemcpy(M_velocity_dev, M_velocity, sizeof(float)*9, hipMemcpyHostToDevice);
					
							fastRayTracingBinMode_rotationSlit(vectorBin, NUMVECTORBIN*PITCHBIN/2.0, PITCHBIN, nv, nv_dev, randomArray_dev, state_dev, value, photonPosition, photonPosition_dev, stack, stack_dev, numStack, workPlace, workPlace_dev, vectorsResult_dev, &accumulatedNum, M_velocity_dev);
						
						}
					}
				}
				
				printf("%d is complete\n", i);
				printf("maximum error is %d\n", numMax);
			}
			fwrite(vectorBin, sizeof(int), NUMVECTORBIN, fid);
			printf("projection %d is finished!\n", idxProjection);
		}
		printf("Angle %d is finished!\n", idxAngle);
		fclose(fid);
	}

	end=std::clock();
	printf("time is %d\n", end-start);
	getchar();
	return 0;
}