#include "hip/hip_runtime.h"
#include"header_simulation_engine.h"

#define NUMPROJECTIONS 1
#define QUEUELENGTH 10							//the size of maximum number of facelets a photon traveling in a straight line can pass
#define DEVIDENT 8								//The striding step in search the result matrix workplace, in order to search faster					
#define NUMPERBATCH 1024
#define NUMVECTORBIN 600
#define PITCHBIN 0.1
#define PHANTOMSIZEX 50
#define PHANTOMSIZEY 50
#define PHANTOMSIZEZ 50
#define PHANTOMPITCHX 0.5
#define PHANTOMPITCHY 0.5
#define PHANTOMPITCHZ 0.5

int main(){
	char *workPlace, *workPlace_dev, buffer[150];
	int idxProjection, numFacelet1, numFacelet2, value, numStack=0, i, j, k, cors[2], numMax=0, *detectorBin;
	float theta, phi, valueFloat, M_source[9], M_velocity[9], *M_velocity_dev, *nv, *nv_dev, photonPosition[3], tempPosition[3], centerOfAperturePosition[3], *photonPosition_dev, angleAlpha=0, distOriginToCA=51.5, rotationSpec[2], EdgePointFinal[3]={0,1,0}, *phantom;
	double *randomArray_dev;
	unsigned long int accumulatedNum=0;
	specFacet* part1,* part2, *stack, *stack_dev;
	vectors *vectorsResult_dev;
	FILE *fid, *fidPhantom;
	std::default_random_engine generator;
	hiprandState *state_dev;

	readPart(&part1, &numFacelet1, "G:\\research\\slitSPECT\\pinholeAperture_assemble50X50_spacing_0.3 - pinholeAperture50X50_0.3dia-1.STL",'t');
	readPart(&part2, &numFacelet2, "G:\\research\\slitSPECT\\pinholeAperture_assemble50X50_spacing_0.3 - detector50X50-1.STL",'d');
	catenate(&stack, &numStack, &part1, & numFacelet1);
	catenate(&stack, &numStack, &part2, & numFacelet2);

	hipMalloc((void**)&stack_dev,sizeof(specFacet)*numStack);
	hipMemcpy(stack_dev, stack, sizeof(specFacet)*numStack, hipMemcpyHostToDevice);		
	nv=(float *)malloc(sizeof(float)*NUMPERBATCH*3);
	hipMalloc((void**)&nv_dev, sizeof(float)*NUMPERBATCH*3);
	hipMalloc((void**)&photonPosition_dev, sizeof(float)*3);
	hipMalloc((void**)&M_velocity_dev, sizeof(float)*9);
	workPlace=(char *)malloc(sizeof(char)*numStack*NUMPERBATCH);
	hipMalloc((void**)&workPlace_dev,sizeof(char)*numStack*NUMPERBATCH);
	hipMalloc((void**)&vectorsResult_dev, sizeof(vectors)*numStack);
	detectorBin=(int *)malloc(sizeof(int)*NUMVECTORBIN*NUMVECTORBIN);
	hipMalloc((void**)&randomArray_dev, sizeof(float)*NUMPERBATCH*2);
	hipMalloc((void**)&state_dev, sizeof(hiprandState)*NUMPERBATCH*2);
	phantom=(float *)malloc(sizeof(float)*PHANTOMSIZEX*PHANTOMSIZEY*PHANTOMSIZEZ);

	std::clock_t start, end;
	start=std::clock();

	theta=PI/4;
	phi=PI/4;

	fid=fopen("G:\\research\\data\\rotatingSlitProjection_pinholeComparisonX0.3X3_90_600_PinholeX1_A1.0_view_45_45.bin","wb");
	fidPhantom=fopen("G:\\research\\data\\phantom.bin","rb");
	fread(phantom, sizeof(float), PHANTOMSIZEX*PHANTOMSIZEY*PHANTOMSIZEZ, fidPhantom);
	fclose(fidPhantom);

	for(idxProjection=0;idxProjection<NUMPROJECTIONS;idxProjection++){		
		centerOfAperturePosition[0]=distOriginToCA*sin(theta)*cos(phi);//30.75;
		centerOfAperturePosition[1]=distOriginToCA*sin(theta)*sin(phi);//30.75;
		centerOfAperturePosition[2]=distOriginToCA*cos(theta);//43.487067043;

		angleAlpha=PI*idxProjection/NUMPROJECTIONS;

		inverseMatrixGeneration(M_source, angleAlpha, centerOfAperturePosition);

		centerOfAperturePosition[0]=0;
		centerOfAperturePosition[1]=0;
		centerOfAperturePosition[2]=distOriginToCA;

		for(i=0;i<NUMVECTORBIN*NUMVECTORBIN;i++){
			detectorBin[i]=0;
		}
		//
		int countTotal = 0;
		for (i = 0; i < PHANTOMSIZEX; i++){
			for (j = 0; j < PHANTOMSIZEY; j++){
				for (k = 0; k < PHANTOMSIZEZ; k++){
					countTotal += phantom[i*PHANTOMSIZEY*PHANTOMSIZEZ + j*PHANTOMSIZEZ + k];
				}
			}
		}
		//
		for(i=0;i<PHANTOMSIZEX;i++){
			for(j=0;j<PHANTOMSIZEY;j++){
				for(k=0;k<PHANTOMSIZEZ;k++){
					valueFloat=phantom[i*PHANTOMSIZEY*PHANTOMSIZEZ+j*PHANTOMSIZEZ+k]*0.905499788919*100;
					if(valueFloat){
						std::poisson_distribution<int> distributionPo(valueFloat);
						value=distributionPo(generator);
					}else{
						value=0;
					}
					if(value){
						tempPosition[0]=i*PHANTOMPITCHX-PHANTOMSIZEX*PHANTOMPITCHX/2.0;
						tempPosition[1]=j*PHANTOMPITCHX-PHANTOMSIZEY*PHANTOMPITCHX/2.0;
						tempPosition[2]=k*PHANTOMPITCHX-PHANTOMSIZEZ*PHANTOMPITCHX/2.0;
						photonPosition[0]=tempPosition[0]*M_source[0]+tempPosition[1]*M_source[1]+tempPosition[2]*M_source[2];
						photonPosition[1]=tempPosition[0]*M_source[3]+tempPosition[1]*M_source[4]+tempPosition[2]*M_source[5];
						photonPosition[2]=tempPosition[0]*M_source[6]+tempPosition[1]*M_source[7]+tempPosition[2]*M_source[8];
						hipMemcpy(photonPosition_dev, photonPosition, sizeof(float)*3, hipMemcpyHostToDevice);

						rotationAngleGeneration(rotationSpec, EdgePointFinal, photonPosition, centerOfAperturePosition);
						velocityRotationMatrixGeneration(M_velocity, rotationSpec, photonPosition, centerOfAperturePosition, 0);
						hipMemcpy(M_velocity_dev, M_velocity, sizeof(float)*9, hipMemcpyHostToDevice);
					
						fastRayTracingBinMode_rotationSlit_pinholeComparison(detectorBin, NUMVECTORBIN*PITCHBIN/2, PITCHBIN, nv, nv_dev, randomArray_dev, state_dev, value, photonPosition, photonPosition_dev, stack, stack_dev, numStack, workPlace, workPlace_dev, vectorsResult_dev, &accumulatedNum, M_velocity_dev);
					}
				}
			}
			printf("%d is complete\n", i);
			printf("maximum error is %d\n", numMax);
		}
		fwrite(detectorBin, sizeof(int), NUMVECTORBIN*NUMVECTORBIN, fid);
		printf("projection %d is finished!\n", idxProjection);
	}
	fclose(fid);

	end=std::clock();
	fclose(fid);	
	
	printf("time is %d\n", end-start);
	getchar();
	return 0;
}